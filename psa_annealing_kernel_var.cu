#include <hip/hip_runtime.h>

extern "C"
{
    __global__ void calculate_cut_val(int vertex, int8_t* J_matrix, int* spin_vector, float* cut_val) 
    {
        int idx = blockIdx.y * blockDim.y + threadIdx.y;
        int stride = blockDim.y * gridDim.y;

        // Use shared memory for intermediate sum reduction
        extern __shared__ float shared_cut[]; 
        shared_cut[threadIdx.y] = 0.0f;

        for (int i = idx; i < vertex; i += stride) {
            for (int j = i + 1; j < vertex; j++) {
                shared_cut[threadIdx.y] -= static_cast<int>(J_matrix[i * vertex + j]) * (1.0f - spin_vector[i] * spin_vector[j]);
            }
        }
        __syncthreads();

        // Perform reduction to sum the values in shared memory
        if (threadIdx.y == 0) {
            float block_sum = 0.0f;
            for (int k = 0; k < blockDim.y; k++) {
                block_sum += shared_cut[k]/2;
            }
            atomicAdd(cut_val, block_sum);
        }  
    }

    __global__ void annealing_module(int vertex, float mem_I0, int8_t *h_vector, int8_t *J_matrix, int *spin_vector, float *rnd, float *lambda, float *delta, int *nu, int count_device)
    {

        int i, k;
        float D_res;

        i = blockIdx.y * blockDim.y + threadIdx.y;
        //j = blockIdx.x * blockDim.x + threadIdx.x; 
        
        if (i < vertex)
        {
            if (count_device % nu[i] == 0)
            {
                    D_res = h_vector[i];
                    __syncthreads();
                    for(k=0; k<vertex; k++){
                        D_res += static_cast<int>(J_matrix[i * vertex + k]) * spin_vector[k];
                    }
                
                    float Itanh = tanh(lambda[i] * mem_I0 * (D_res + delta[i]))  + rnd[i];
                    spin_vector[i] = (Itanh > 0) ? 1 : -1;
                    
                    __syncthreads();

            }
            
        }
    }
}
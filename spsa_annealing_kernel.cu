#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C"
{
    __global__ void calculate_cut_val(int vertex, int8_t* J_matrix, int* spin_vector, float* cut_val) 
    {
        int idx = blockIdx.y * blockDim.y + threadIdx.y;
        int stride = blockDim.y * gridDim.y;

        // Use shared memory for intermediate sum reduction
        extern __shared__ float shared_cut[]; 
        shared_cut[threadIdx.y] = 0.0f;

        for (int i = idx; i < vertex; i += stride) {
            for (int j = i + 1; j < vertex; j++) {
                shared_cut[threadIdx.y] -= static_cast<int>(J_matrix[i * vertex + j]) * (1.0f - spin_vector[i] * spin_vector[j]);
            }
        }
        __syncthreads();

        // Perform reduction to sum the values in shared memory
        if (threadIdx.y == 0) {
            float block_sum = 0.0f;
            for (int k = 0; k < blockDim.y; k++) {
                block_sum += shared_cut[k]/2;
            }
            atomicAdd(cut_val, block_sum);
        }  
    }

    __global__ void annealing_module(float stall_prop, int vertex, float mem_I0, int8_t *h_vector, int8_t *J_matrix, int *spin_vector, float *rnd)
    {

        int i, k;
        float D_res;

        i = blockIdx.y * blockDim.y + threadIdx.y;
        
        // curandStateを初期化
        hiprandState state;
        hiprand_init((unsigned long long)clock() + i, 0, 0, &state);

        if (i < vertex)
        {
            D_res = h_vector[i];
            __syncthreads();
            for (k = 0; k < vertex; k++)
            {
                D_res += static_cast<int>(J_matrix[i * vertex + k]) * spin_vector[k];
            }
            
            float Itanh = tanh(mem_I0 * D_res) + rnd[i];;
            
            // 乱数を生成してstall_propと比較
            float rand_val = hiprand_uniform(&state);
            if (rand_val >= stall_prop)
            {
                spin_vector[i] = (Itanh > 0) ? 1 : -1;
            }
            
            __syncthreads();
        }
    }
}
